#include <stdio.h>
#include <stdlib.h>
#include "mandelgpu.h"
#include "writepng.h"
#include <omp.h>
#include <hip/hip_runtime_api.h>
#define NUM_THREADS 1
#define NUM_BLOCKS 1

int main(int argc, char *argv[])
{

    int width, height;
    int max_iter;
    int *h_image, *d_image;
    double ts, te;

    width = 4096;
    height = 4096;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if (argc == 2)
        width = height = atoi(argv[1]);

    // Allocate memory on host and device
    hipHostMalloc((void **)&h_image, width * height * sizeof(int));
    hipMalloc((void **)&d_image, width * height * sizeof(int));

    if (d_image == NULL || h_image == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return (1);
    }

    // Start timer
    ts = omp_get_wtime();

    // Launch kernel (only using 1 thread ATM)
    dim3 threadsPerBlock(64,64); // 64*64 = 4096 threads in total (1 per pixel)
    mandel<<<NUM_BLOCKS, threadsPerBlock>>>(width, height, d_image, max_iter);
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(h_image, d_image, width * height * sizeof(int), hipMemcpyDeviceToHost);

    // End timer
    te = omp_get_wtime() - ts;

    // Save pic
    writepng("mandelbrot.png", h_image, width, height);

    printf("%f\n", te);

    // Cleanup
    hipHostFree(h_image);
    hipFree(d_image);

    return (0);
}
