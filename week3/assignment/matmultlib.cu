#include "hip/hip_runtime.h"
extern "C" {
#include <cblas.h>
#include <math.h>
}
#include "matmultgpu.h"
#include <hip/hip_runtime_api.h>

extern "C" {
void matmult_nat(int m, int n, int k, double *A, double *B, double *C)
{
    // Initializing C
    int i, j;
    for (i = 0; i < m; i++)
        for (j = 0; j < n; j++)
            C[i * n + j] = 0.0;

    // Perform multiplication A*B
    int t, q;
    for (i = 0; i < m; i++)
        for (j = 0; j < n; j++)
            for (q = 0, t = 0; t < k && q < k; t++, q++)
                C[i * n + j] += A[i * k + q] * B[t * n + j];
}

void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}

void matmult_gpu1(int m, int n, int k, double *h_A, double *h_B, double *h_C)
{
    double *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    if (d_A == NULL || d_B == NULL || d_C == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    matmult_kernel_gpu1<<<1,1>>>(d_A,d_B,d_C,m,n,k);
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void matmult_gpu2(int m, int n, int k, double *h_A, double *h_B, double *h_C)
{
    double *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    if (d_A == NULL || d_B == NULL || d_C == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threads_per_block(16, 16); // e.g. 16*16 = 256 threads in total
    dim3 num_blocks(ceil((double)n/threads_per_block.y),ceil((double)m/threads_per_block.x));
    matmult_kernel_gpu2<<<num_blocks,threads_per_block>>>(d_A,d_B,d_C,m,n,k);
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void matmult_gpu3(int m, int n, int k, double *h_A, double *h_B, double *h_C)
{
    double *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    if (d_A == NULL || d_B == NULL || d_C == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threads_per_block(16, 16); // e.g. 16*16 = 256 threads in total
    dim3 num_blocks(ceil((double)n/(threads_per_block.x)),ceil((double)m/(threads_per_block.y*2)));
    matmult_kernel_gpu3<<<num_blocks,threads_per_block>>>(d_A,d_B,d_C,m,n,k);
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
}