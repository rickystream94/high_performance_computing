#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "matrixlib.h"
#include <math.h>
#include "jacobigpu.h"
#define DOMAIN_LENGTH 2
#define BLOCK_SIZE 16

void jacobi_1(int N, int k_max)
{
    // Variables declaration
    int k;
    double *h_U, *h_f, *d_U, *d_U_old, *d_U_new, *d_f, *temp_ptr;
    double delta, ts, te;

    // 2. allocate memory for the necessary data fields
    hipMalloc((void **)&d_U, N * N * sizeof(double));
    hipMalloc((void **)&d_U_new, N * N * sizeof(double));
    hipMalloc((void **)&d_f, N * N * sizeof(double));
    hipHostMalloc((void **)&h_U, N * N * sizeof(double));
    hipHostMalloc((void **)&h_f, N * N * sizeof(double));

    // Calculate delta
    delta = (double)(DOMAIN_LENGTH) / (N - 1);

    // 3. initialize the fields with your start and boundary conditions
    init_f_matrix(N, h_f, delta);
    init_U_matrix(N, h_U);

    // 4. copy data from host to device
    hipMemcpy(d_U,h_U,N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_f,h_f,N * N * sizeof(double), hipMemcpyHostToDevice);

    // 5. call kernel iterator
    // Get starting time
    ts = omp_get_wtime();

    // Assign mat_old with the initial guess (k = 0 iteration)
    d_U_old = d_U;
    for(k = 0; k < k_max; k++)
    {
        jacobi_kernel1<<<1,1>>>(N, d_U_old, d_U_new, d_f, delta);
        checkCudaErrors(hipDeviceSynchronize());

        // Swap the pointers on the CPU
        {
            temp_ptr = d_U_old;
            d_U_old = d_U_new;
            d_U_new = temp_ptr;
        }
    }
    
    // Get ending time
    te = omp_get_wtime() - ts;

    // Copy result back to host (notice that d_U_old will have the last good result!)
    hipMemcpy(h_U, d_U_old, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // 6. print results, e.g. timings, data, etc
    //print_matrix(N, h_U); // Only for debugging
    printf("%f\n", te);

    // 7. Cleanup
    hipHostFree(h_U);
    hipHostFree(h_f);
    hipFree(d_f);
    hipFree(d_U);
    hipFree(d_U_new);
}

void jacobi_2(int N, int k_max)
{
    // Variables declaration
    int k;
    double *h_U, *h_f, *d_U, *d_U_old, *d_U_new, *d_f, *temp_ptr;
    double delta, ts, te;

    // 2. allocate memory for the necessary data fields
    hipMalloc((void **)&d_U, N * N * sizeof(double));
    hipMalloc((void **)&d_U_new, N * N * sizeof(double));
    hipMalloc((void **)&d_f, N * N * sizeof(double));
    hipHostMalloc((void **)&h_U, N * N * sizeof(double));
    hipHostMalloc((void **)&h_f, N * N * sizeof(double));

    // Calculate delta
    delta = (double)(DOMAIN_LENGTH) / (N - 1);

    // 3. initialize the fields with your start and boundary conditions
    init_f_matrix(N, h_f, delta);
    init_U_matrix(N, h_U);

    // 4. copy data from host to device
    hipMemcpy(d_U,h_U,N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_f,h_f,N * N * sizeof(double), hipMemcpyHostToDevice);

    // Define GPU thread blocks dimensions
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE); // e.g. 16*16 = 256 threads in total
    dim3 num_blocks(ceil((double)N/threads_per_block.x), ceil((double)N/threads_per_block.y));

    // 5. call kernel iterator
    // Get starting time
    ts = omp_get_wtime();

    // Assign mat_old with the initial guess (k = 0 iteration)
    d_U_old = d_U;
    for(k = 0; k < k_max; k++)
    {
        jacobi_kernel2<<<num_blocks,threads_per_block>>>(N, d_U_old, d_U_new, d_f, delta);
        checkCudaErrors(hipDeviceSynchronize());

        // Swap the pointers on the CPU
        {
            temp_ptr = d_U_old;
            d_U_old = d_U_new;
            d_U_new = temp_ptr;
        }
    }
    
    // Get ending time
    te = omp_get_wtime() - ts;

    // Copy result back to host (notice that d_U_old will have the last good result!)
    hipMemcpy(h_U, d_U_old, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // 6. print results, e.g. timings, data, etc
    //print_matrix(N, h_U); // Only for debugging
    printf("%f\n", te);

    // 7. Cleanup
    hipHostFree(h_U);
    hipHostFree(h_f);
    hipFree(d_f);
    hipFree(d_U);
    hipFree(d_U_new);
}

void jacobi_3(int N, int k_max)
{
    // Variables declaration
    int k;
    double *h_U, *h_f, *d0_U, *d1_U, *d0_U_old, *d1_U_old, *d0_U_new, *d1_U_new, *d0_f, *d1_f, *temp_ptr;
    double delta, ts, te;

    // Allocate host memory
    hipHostMalloc((void **)&h_U, N * N * sizeof(double));
    hipHostMalloc((void **)&h_f, N * N * sizeof(double));

    // Calculate delta
    delta = (double)(DOMAIN_LENGTH) / (N - 1);

    // 3. initialize the fields with your start and boundary conditions
    init_f_matrix(N, h_f, delta);
    init_U_matrix(N, h_U);

    // Define GPU thread blocks dimensions
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE); // e.g. 16*16 = 256 threads in total
    dim3 num_blocks(ceil((double)N/(threads_per_block.x*2)), ceil((double)N/(threads_per_block.y*2)));

    // DEVICE 0
    hipSetDevice(0);
    // 2. allocate memory for the necessary data fields
    hipMalloc((void **)&d0_U, N/2 * N * sizeof(double));
    hipMalloc((void **)&d0_U_new, N/2 * N * sizeof(double));
    hipMalloc((void **)&d0_f, N/2 * N * sizeof(double));

    // 4. copy data from host to device
    hipMemcpy(d0_U,h_U,N/2 * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d0_f,h_f,N/2 * N * sizeof(double), hipMemcpyHostToDevice);

    // DEVICE 1
    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);
    // 2. allocate memory for the necessary data fields
    hipMalloc((void **)&d1_U, N/2 * N * sizeof(double));
    hipMalloc((void **)&d1_U_new, N/2 * N * sizeof(double));
    hipMalloc((void **)&d1_f, N/2 * N * sizeof(double));

    // 4. copy data from host to device
    hipMemcpy(d1_U,h_U + N/2,N/2 * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d1_f,h_f + N/2,N/2 * N * sizeof(double), hipMemcpyHostToDevice);

    // Get starting time
    ts = omp_get_wtime();

    // Assign mat_old with the initial guess (k = 0 iteration)
    d0_U_old = d0_U;
    d1_U_old = d1_U;
    for(k = 0; k < k_max; k++)
    {
        hipSetDevice(0);
        jacobi_kernel_multigpu_0<<<num_blocks,threads_per_block>>>(N, d0_U_old, d0_U_new, d0_f, delta);
        hipSetDevice(1);
        jacobi_kernel_multigpu_1<<<num_blocks,threads_per_block>>>(N, d1_U_old, d1_U_new, d1_f, delta);
        checkCudaErrors(hipDeviceSynchronize());

        // Swap the pointers on the CPU
        temp_ptr = d0_U_old;
        d0_U_old = d0_U_new;
        d0_U_new = temp_ptr;

        temp_ptr = d1_U_old;
        d1_U_old = d1_U_new;
        d1_U_new = temp_ptr;
    }
    
    // Get ending time
    te = omp_get_wtime() - ts;

    // Copy result back to host (notice that d_U_old will have the last good result!)
    hipMemcpy(h_U, d0_U_old, N/2 * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_U + N/2, d1_U_old, N/2 * N * sizeof(double), hipMemcpyDeviceToHost);

    // 6. print results, e.g. timings, data, etc
    //print_matrix(N, h_U); // Only for debugging
    printf("%f\n", te);

    // 7. Cleanup
    hipHostFree(h_U);
    hipHostFree(h_f);
    hipSetDevice(0);
    hipFree(d0_f);
    hipFree(d0_U);
    hipFree(d0_U_new);
    hipSetDevice(1);
    hipFree(d1_f);
    hipFree(d1_U);
    hipFree(d1_U_new);
}